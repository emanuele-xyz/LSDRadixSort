#include "hip/hip_runtime.h"
#include <iostream>
#include <stdint.h>

// TODO: Test code in release build! 
// It looks like LSDBinaryRadixSort doesn't work in release build
// This is probably due to a race condition
// Sprinkle code with __syncthreads() and check what happens

//#define BENCHMARK_BUILD_HISTOGRAMS
//#define BENCHMARK_GPU_LSD_RADIX_SORT
#define PRINT_TIMINGS

/*
	TODO: watch this https://www.youtube.com/watch?v=fsC3QeZHM1U
*/

// Windows only
#define _CRTDBG_MAP_ALLOC
#include <stdlib.h>
#include <crtdbg.h>
#include <intrin.h>

#include "Utils.h"
#include "CudaUtils.h"

void LSDRadixSortPass(uint32_t* in, uint32_t* out, int count, uint32_t* histogram, int r, int bit_group)
{
	memset(histogram, 0, sizeof(*histogram) * (1 << r));

	// build histogram
	for (int j = 0; j < count; j++)
	{
		uint32_t val = in[j];
		int key = GET_R_BITS(val, r, bit_group);
		histogram[key]++;
	}

	// prefix sum of histogram
	for (int j = 1; j < (1 << r); j++)
	{
		histogram[j] += histogram[j - 1];
	}

	// permute input elements into output
	for (int j = count - 1; j >= 0; j--)
	{
		uint32_t val = in[j];
		int key = GET_R_BITS(val, r, bit_group);
		histogram[key]--;
		out[histogram[key]] = val;
	}

	// copy output array into input array
	memcpy(in, out, sizeof(*in) * count);
}

/*
* in:    input array (will be modified)
* out:   output array
* count: number of elements to sort
* r:     number of bits to consider as keys (any factor of 32 apart from itself)
*/
void LSDRadixSort(uint32_t* in, uint32_t* out, int count, uint32_t* histogram, int r)
{
	int bit_groups = (sizeof(*in) * 8) / r;
	for (int bit_group = 0; bit_group < bit_groups; bit_group++)
	{
		LSDRadixSortPass(in, out, count, histogram, r, bit_group);
	}
}

void PrefixSum(uint32_t* a, int count)
{
	for (int i = 1; i < count; i++)
	{
		a[i] += a[i - 1];
	}
	for (int i = count - 1; i >= 1; i--)
	{
		a[i] = a[i - 1];
	}
	a[0] = 0;
}

__device__ void SMEMUpSweep(volatile uint32_t* smem, int bdim, int tid)
{
	for (int d = 0; (1 << d) < bdim; d++)
	{
		int offset = (1 << (d + 1));
		int bias = offset - 1;
		int shift = (1 << d);

		if (tid < (bdim >> (d + 1)))
		{
			int index = bias + tid * offset;
			int left = index - shift;
			smem[index] += smem[left];
		}
		__syncthreads();
	}
}

__device__ void SMEMDownSweep(volatile uint32_t* smem, int bdim, int tid)
{
	for (int d = 0; (1 << d) < bdim; d++)
	{
		int offset = (bdim >> d);
		int bias = offset - 1;
		int shift = (bdim >> (d + 1));

		if (tid < (1 << d))
		{
			int index = bias + tid * offset;
			int left = index - shift;
			int l = smem[index];
			int r = smem[index] + smem[left];
			smem[left] = l;
			smem[index] = r;
		}
		__syncthreads();
	}
}

__global__ void BlockPrefixSumKernel(uint32_t* a, uint32_t* block_sums)
{
	extern __shared__ uint32_t smem[];

	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int bdim = blockDim.x;
	int i = bid * bdim + tid;

	// load array into smem
	smem[tid] = a[i];
	__syncthreads();

	SMEMUpSweep(smem, bdim, tid);
	if (tid == 0)
	{
		if (block_sums)
		{
			block_sums[bid] = smem[bdim - 1];
		}
		smem[bdim - 1] = 0;
	}
	__syncthreads();
	SMEMDownSweep(smem, bdim, tid);

	// write smem into array
	a[i] = smem[tid];
}

#define SEQ_RADIX_SORT_TEST_ELEMS_COUNT (1024 * 4)
#define SEQ_RADIX_SORT_TEST_ELEMS_MIN 0
#define SEQ_RADIX_SORT_TEST_ELEMS_MAX UINT32_MAX
#define SEQ_RADIX_SORT_TEST_ELEMS_R 4

void TestSequentialLSDRadixSort()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test sequential LSD radix sort --" << std::endl;
	#endif

	RNG rng = RNG(0, SEQ_RADIX_SORT_TEST_ELEMS_MIN, SEQ_RADIX_SORT_TEST_ELEMS_MAX);

	int count = SEQ_RADIX_SORT_TEST_ELEMS_COUNT;
	uint32_t* a = (uint32_t*)(calloc(count, sizeof(*a)));
	uint32_t* b = (uint32_t*)(calloc(count, sizeof(*b)));
	uint32_t* c = (uint32_t*)(calloc(count, sizeof(*c)));
	uint32_t* h = (uint32_t*)(calloc(1 << SEQ_RADIX_SORT_TEST_ELEMS_R, sizeof(*h)));

	// populate a and c
	for (int i = 0; i < count; i++)
	{
		int elem = rng.Get();
		a[i] = elem;
		c[i] = elem;
	}

	// sort a writing result in b using LSD radix sort
	float lsd_radix_sort_ms = 0;
	{
		int64_t start = GetTimestamp();
		LSDRadixSort(a, b, count, h, SEQ_RADIX_SORT_TEST_ELEMS_R);
		int64_t end = GetTimestamp();
		lsd_radix_sort_ms = GetElapsedMS(start, end);
	}

	// sort c using standard library sort
	float std_sort_ms = 0;
	{
		int64_t start = GetTimestamp();
		std::sort(c, c + count);
		int64_t end = GetTimestamp();
		std_sort_ms = GetElapsedMS(start, end);
	}

	#ifdef PRINT_ARRAY
	PrintArray('b', b, count);
	PrintArray('c', c, count);
	#endif

	#ifdef PRINT_TIMINGS
	std::cout << "Sequential LSD Radix Sort: " << lsd_radix_sort_ms << " ms" << std::endl;
	std::cout << "Sequential STD Sort: " << std_sort_ms << " ms" << std::endl;
	std::cout << "Speedup: x" << std_sort_ms / lsd_radix_sort_ms << std::endl;
	#endif

	CheckArrays(b, c, count);

	free(h);
	free(c);
	free(b);
	free(a);
}

#define BLOCK_PREFIX_SUM_TEST_ELEMS_COUNT (1024)
#define BLOCK_PREFIX_SUM_TEST_ELEMS_MIN 0
#define BLOCK_PREFIX_SUM_TEST_ELEMS_MAX 10

void TestBlockPrefixSumKernel()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test block exclusive prefix sum --" << std::endl;
	#endif

	RNG rng = RNG(0, BLOCK_PREFIX_SUM_TEST_ELEMS_MIN, BLOCK_PREFIX_SUM_TEST_ELEMS_MAX);

	int count = BLOCK_PREFIX_SUM_TEST_ELEMS_COUNT;
	size_t size = count * sizeof(uint32_t);
	size_t blocks_size = 1 * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_block_sums = (uint32_t*)MyCudaMalloc(blocks_size);

	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	float parallel_ms = 0;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		BlockPrefixSumKernel << <1, count, size >> > (d_a, d_block_sums);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost));
		parallel_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	float sequential_ms = 0;
	{
		int64_t start = GetTimestamp();
		PrefixSum(h_a, count);
		int64_t end = GetTimestamp();
		sequential_ms = GetElapsedMS(start, end);
	}

	#ifdef PRINT_ARRAY
	PrintArray('a', h_a, count);
	PrintArray('b', h_b, count);
	#endif

	#ifdef PRINT_TIMINGS
	std::cout << "Prefix Sum Sequential: " << sequential_ms << " ms" << std::endl;
	std::cout << "Prefix Sum Block: " << parallel_ms << " ms" << std::endl;
	std::cout << "Speedup: x" << sequential_ms / parallel_ms << std::endl;
	#endif

	CheckArrays(h_a, h_b, count);

	CUDA_CALL(hipFree(d_block_sums));
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_b));
	CUDA_CALL(hipHostFree(h_a));
}

int GetGPUPrefixSumBlockSumsCount(int count, int threads_per_block)
{
	int total_block_sums_count = 0;
	while (count > threads_per_block)
	{
		MYASSERT(count % threads_per_block == 0);
		int block_sums_count = count / threads_per_block;
		total_block_sums_count += block_sums_count;
		count = block_sums_count;
	}
	return total_block_sums_count + 1;
}

__global__ void AddBlockSumsKernel(uint32_t* a, uint32_t* block_sums)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int i = blockDim.x * bid + tid;
	a[i] += block_sums[bid];
}

void GPUPrefixSum(uint32_t* d_a, int count, int threads_per_block, uint32_t* d_block_sums, hipStream_t s = 0)
{
	if (count <= threads_per_block)
	{
		int smem = count * sizeof(uint32_t);
		BlockPrefixSumKernel << <1, count, smem, s >> > (d_a, d_block_sums);
	}
	else
	{
		int smem = threads_per_block * sizeof(uint32_t);
		int blocks = count / threads_per_block;
		BlockPrefixSumKernel << <blocks, threads_per_block, smem, s >> > (d_a, d_block_sums);
		GPUPrefixSum(d_block_sums, blocks, threads_per_block, &d_block_sums[blocks], s);
		// Skip the first block, since the exclusive block sum will have 0
		AddBlockSumsKernel << <blocks - 1, threads_per_block, 0, s >> > (&d_a[threads_per_block], &d_block_sums[1]);
	}
}

void TestGPUPrefixSum(int count, int threads_per_block, int min, int max)
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test exclusive prefix sum --" << std::endl;
	#endif

	RNG rng = RNG(0, min, max);

	// allocate
	int block_sums_count = GetGPUPrefixSumBlockSumsCount(count, threads_per_block);
	size_t size = count * sizeof(uint32_t);
	size_t block_sums_size = block_sums_count * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_block_sums = (uint32_t*)MyCudaMalloc(block_sums_size);

	// populate input
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	// parallel prefix sum
	float parallel_ms = 0;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		GPUPrefixSum(d_a, count, threads_per_block, d_block_sums);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost));
		parallel_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// sequential prefix sum
	float sequential_ms = 0;
	{
		int64_t start = GetTimestamp();
		PrefixSum(h_a, count);
		int64_t end = GetTimestamp();
		sequential_ms = GetElapsedMS(start, end);
	}

	// print arrays
	#ifdef PRINT_ARRAY
	PrintArray('a', h_a, count);
	PrintArray('b', h_b, count);
	#endif

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "Prefix sum of " << (double)(size) / (1024.0 * 1024.0 * 1024.0) << " GB of data" << std::endl;
	std::cout << "Threads per block: " << threads_per_block << std::endl;
	std::cout << "Prefix Sum Sequential: " << sequential_ms << " ms" << std::endl;
	std::cout << "GPU Prefix Sum: " << parallel_ms << " ms" << std::endl;
	std::cout << "Speedup: x" << sequential_ms / parallel_ms << std::endl;
	#endif

	// check arrays
	CheckArrays(h_a, h_b, count);

	// deallocate
	CUDA_CALL(hipFree(d_block_sums));
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_b));
	CUDA_CALL(hipHostFree(h_a));
}

__device__ void SMEMLSDBinaryRadixSort(volatile uint32_t* a, int tid, int bdim, int first_bit = 0, int bit_count = 32)
{
	// passes
	for (int i = first_bit; i < first_bit + bit_count; i++)
	{
		uint32_t val = a[tid];
		uint32_t bit = GET_R_BITS(val, 1, i);
		// invert bit and write it to a
		a[tid] = bit ? 0 : 1;
		__syncthreads();

		// prefix sum of inverted bits
		SMEMUpSweep(a, bdim, tid);
		uint32_t total_falses = a[bdim - 1];
		__syncthreads(); // NOTE: don't know why, but without it, release build breaks
		if (tid == 0) a[bdim - 1] = 0;
		__syncthreads();
		SMEMDownSweep(a, bdim, tid);

		// now a holds the destination for false keys
		// destination for true key
		uint32_t t = (uint32_t)tid - a[tid] + total_falses;
		// destination for val
		uint32_t d = bit ? t : a[tid];
		__syncthreads();
		// write val to destination
		a[d] = val;
		__syncthreads();
	}
}

__global__ void LSDBinaryRadixSortKernel(uint32_t* a, int first_bit = 0, int bit_count = 32)
{
	extern __shared__ uint32_t smem[];

	int tid = threadIdx.x;
	int bdim = blockDim.x;
	int bid = blockIdx.x;
	int idx = bid * bdim + tid;

	// load array in smem
	smem[tid] = a[idx];
	__syncthreads();

	SMEMLSDBinaryRadixSort(smem, tid, blockDim.x, first_bit, bit_count);

	// write smem into array
	a[idx] = smem[tid];
}

#define LSD_BINARY_RADIX_SORT_TEST_ELEMS_COUNT (1024)
#define LSD_BINARY_RADIX_SORT_TEST_ELEMS_MIN 0
#define LSD_BINARY_RADIX_SORT_TEST_ELEMS_MAX UINT32_MAX

void TestLSDBinaryRadixSort()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test LSD Binary Radix Sort --" << std::endl;
	#endif

	RNG rng = RNG(0, LSD_BINARY_RADIX_SORT_TEST_ELEMS_MIN, LSD_BINARY_RADIX_SORT_TEST_ELEMS_MAX);

	// allocate
	int count = LSD_BINARY_RADIX_SORT_TEST_ELEMS_COUNT;
	size_t size = count * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);

	// populate input
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	// parallel lsd binary radix sort
	float parallel_lsd_binary_radix_sort_ms = 0;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		LSDBinaryRadixSortKernel << <1, count, size >> > (d_a);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost));
		parallel_lsd_binary_radix_sort_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// sequential sort
	float std_sort_ms = 0;
	{
		int64_t start = GetTimestamp();
		std::sort(h_a, h_a + count);
		int64_t end = GetTimestamp();
		std_sort_ms = GetElapsedMS(start, end);
	}

	// print arrays
	#ifdef PRINT_ARRAY
	PrintArray('a', h_a, count);
	PrintArray('b', h_b, count);
	#endif

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "STD Sort: " << std_sort_ms << " ms" << std::endl;
	std::cout << "Parallel LSD Binary Radix Sort: " << parallel_lsd_binary_radix_sort_ms << " ms" << std::endl;
	#endif

	// check arrays
	CheckArrays(h_a, h_b, count);

	// deallocate
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_b));
	CUDA_CALL(hipHostFree(h_a));
}

/*
	a: m x n matrix
	b: n x m matrix
*/
void Transpose(uint32_t* a, uint32_t* b, int m, int n)
{
	for (int row = 0; row < m; row++)
	{
		for (int col = 0; col < n; col++)
		{
			int a_i = row * n + col;
			int b_i = col * m + row;
			b[b_i] = a[a_i];
		}
	}
}

/*
	a : m x n matrix
	b : n x m matrix
*/
__global__ void TransposeNaiveKernel(uint32_t* a, uint32_t* b, int m, int n)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= m || j >= n) return;
	b[j * m + i] = a[i * n + j];
}

/*
	a : m x n matrix
	b : n x m matrix
*/
__global__ void TransposeSMEMKernel(uint32_t* a, uint32_t* b, int m, int n)
{
	extern __shared__ uint32_t smem[];

	int t_i = threadIdx.y;
	int t_j = threadIdx.x;
	int t_cols = blockDim.x;
	int t_rows = blockDim.y;
	int a_i = blockIdx.y * blockDim.y + t_i;
	int a_j = blockIdx.x * blockDim.x + t_j;
	int b_i = blockIdx.x * blockDim.x + t_i;
	int b_j = blockIdx.y * blockDim.y + t_j;

	// copy matrix block into smem
	if (a_i < m && a_j < n)
	{
		smem[t_i * t_cols + t_j] = a[a_i * n + a_j];
	}
	__syncthreads();

	// write transposed smem block
	if (b_i < n && b_j < m)
	{
		b[b_i * m + b_j] = smem[t_j * t_rows + t_i];
	}
}

#define TRANSPOSE_TEST_M (128)
#define TRANSPOSE_TEST_N (32)
#define TRANSPOSE_TEST_BLOCK_DIM 32
#define TRANSPOSE_TEST_MIN_ELEM 0
#define TRANSPOSE_TEST_MAX_ELEM 9

void TestTranspose()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test Transpose --" << std::endl;
	#endif

	RNG rng = RNG(0, TRANSPOSE_TEST_MIN_ELEM, TRANSPOSE_TEST_MAX_ELEM);

	// allocate
	int m = TRANSPOSE_TEST_M;
	int n = TRANSPOSE_TEST_N;
	int count = m * n;
	int block_dim = TRANSPOSE_TEST_BLOCK_DIM;
	size_t size = count * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_c = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_b = (uint32_t*)MyCudaMalloc(size);

	#ifdef PRINT_TIMINGS
	std::cout << "Transpose of " << (double)(size) / (1024.0 * 1024.0 * 1024.0) << " GB of data" << std::endl;
	#endif

	// populate input
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	// sequential transpose
	float sequential_ms = 0.0f;
	{
		int64_t start = GetTimestamp();
		Transpose(h_a, h_b, m, n);
		int64_t end = GetTimestamp();
		sequential_ms = GetElapsedMS(start, end);
	}

	float gpu_naive_ms = 0.0f;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		dim3 block(block_dim, block_dim);
		dim3 grid((n + block_dim - 1) / block_dim, (m + block_dim - 1) / block_dim);
		TransposeNaiveKernel << <grid, block >> > (d_a, d_b, m, n);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_c, d_b, size, hipMemcpyDeviceToHost));
		gpu_naive_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "Sqeuential Transpose: " << sequential_ms << " ms" << std::endl;
	std::cout << "GPU Naive Transpose: " << gpu_naive_ms << " ms - Speedup: x" << sequential_ms / gpu_naive_ms << std::endl;
	#endif

	// check matrices
	CheckArrays(h_b, h_c, count);

	float gpu_smem_ms = 0.0f;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		dim3 block(block_dim, block_dim);
		dim3 grid((n + block_dim - 1) / block_dim, (m + block_dim - 1) / block_dim);
		size_t smem = block_dim * block_dim * sizeof(uint32_t);
		TransposeSMEMKernel << <grid, block, smem >> > (d_a, d_b, m, n);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_c, d_b, size, hipMemcpyDeviceToHost));
		gpu_smem_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "GPU SMEM Transpose: " << gpu_smem_ms << " ms - Speedup: x" << sequential_ms / gpu_smem_ms << std::endl;
	#endif

	// check matrices
	CheckArrays(h_b, h_c, count);

	// deallocate
	hipFree(d_b);
	hipFree(d_a);
	hipHostFree(h_c);
	hipHostFree(h_b);
	hipHostFree(h_a);
}

/*
	grid:  how many histograms
	block: how many array elements per histogram
*/
void BuildHistogramsCPU(uint32_t* a, uint32_t* h, int count, int r, int bit_group, int grid, int block)
{
	int h_count = (1 << r);

	for (int g = 0; g < grid; g++)
	{
		for (int b = 0; b < block; b++)
		{
			int a_i = g * block + b;
			uint32_t val = a[a_i];
			int key = GET_R_BITS(val, r, bit_group);
			int h_i = g * h_count + key;
			h[h_i] += 1;
		}
	}
}

__global__ void BuildHistogramsKernel(uint32_t* a, uint32_t* h, int count, int r, int bit_group)
{
	extern __shared__ uint32_t smem[];

	int bdim = blockDim.x;
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int idx = bid * bdim + tid;

	// zero initialize smem
	int h_count = 1 << r;
	float cells_per_thread_ratio = (float)(h_count) / (float)(bdim);
	int cells_per_thread = cells_per_thread_ratio < 1.0f ? 1 : (int)(cells_per_thread_ratio + 0.5f);
	for (int i = 0; i < cells_per_thread; i++)
	{
		int smem_i = tid * cells_per_thread + i;
		if (smem_i < h_count)
		{
			smem[smem_i] = 0;
		}
	}
	__syncthreads();

	// build histogram in smem
	if (idx < count)
	{
		uint32_t val = a[idx];
		int key = GET_R_BITS(val, r, bit_group);
		atomicInc(&smem[key], UINT32_MAX);
	}
	__syncthreads();

	// write histogram to global memory
	for (int i = 0; i < cells_per_thread; i++)
	{
		int smem_i = tid * cells_per_thread + i;
		int h_i = bid * h_count + smem_i;
		if (smem_i < h_count)
		{
			h[h_i] = smem[smem_i];
		}
	}
}

void TestBuildHistogram(int count, int block, int r, int bit_group, int min, int max)
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test Build Histogram --" << std::endl;
	#endif

	int grid = (count + block - 1) / block;
	size_t h_count = (1 << r);
	size_t h_total_count = h_count * grid;

	// get sizes
	size_t size = count * sizeof(uint32_t);
	size_t h_total_size = h_total_count * sizeof(uint32_t);

	#ifdef PRINT_TIMINGS
	std::cout << "Elements: " << (double)(size) / (1024.0 * 1024.0 * 1024.0) << " GB" << std::endl;
	std::cout << "Histograms: " << (double)(h_total_size) / (1024.0 * 1024.0 * 1024.0) << " GB" << std::endl;
	std::cout << "Block Size: " << block << std::endl;
	std::cout << "R: " << r << std::endl;
	std::cout << "Bit Group: " << bit_group << std::endl;
	#endif

	if (h_total_size > size)
	{
		#ifdef PRINT_TIMINGS
		std::cout << "SKIP: histogram is bigger than input" << std::endl;
		#endif

		#ifdef BENCHMARK_BUILD_HISTOGRAMS
		return;
		#else
		MYCRASH();
		#endif
	}

	// allocate
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_h1 = (uint32_t*)MyCudaHostAlloc(h_total_size);
	uint32_t* h_h2 = (uint32_t*)MyCudaHostAlloc(h_total_size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_h = (uint32_t*)MyCudaMalloc(h_total_size);

	// populate input
	RNG rng = RNG(0, min, max);
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	// cpu implementation
	float cpu_ms = 0.0f;
	{
		int64_t start = GetTimestamp();
		BuildHistogramsCPU(h_a, h_h1, count, r, bit_group, grid, block);
		int64_t end = GetTimestamp();
		cpu_ms = GetElapsedMS(start, end);
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "CPU " << cpu_ms << " ms" << std::endl;
	#endif

	// gpu implementation
	float gpu_ms = 0.0f;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		size_t smem = h_count * sizeof(uint32_t);
		BuildHistogramsKernel << <grid, block, smem >> > (d_a, d_h, count, r, bit_group);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_h2, d_h, h_total_size, hipMemcpyDeviceToHost));
		gpu_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "GPU " << gpu_ms << " ms - Speedup: x" << cpu_ms / gpu_ms << std::endl;
	#endif

	// check arrays
	CheckArrays(h_h1, h_h2, h_total_count);

	// deallocate
	CUDA_CALL(hipFree(d_h));
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_h2));
	CUDA_CALL(hipHostFree(h_h1));
	CUDA_CALL(hipHostFree(h_a));
}

void BenchmarkBuildHistogram()
{
	int counts[] =
	{
		1024 * 1024 * 64,
		1024 * 1024 * 128,
		1024 * 1024 * 256,
		1024 * 1024 * 512,
		1024 * 1024 * 1024,
	};

	int blocks[] =
	{
		64,
		128,
		256,
		512,
		1024,
	};

	int rs[] =
	{
		1,
		2,
		4,
		8,
		16,
	};

	for (int c_i = 0; c_i < MYARRAYCOUNT(counts); c_i++)
	{
		for (int b_i = 0; b_i < MYARRAYCOUNT(blocks); b_i++)
		{
			for (int r_i = 0; r_i < MYARRAYCOUNT(rs); r_i++)
			{
				RNG rng = RNG(0, 0, (32 / rs[r_i]));
				TestBuildHistogram(counts[c_i], blocks[b_i], rs[r_i], rng.Get(), 0, UINT32_MAX);
			}
		}
	}
}

#if 0
#define BUILD_DESTINATION_TABLE_TEST_COUNT (16)
#define BUILD_DESTINATION_TABLE_TEST_BLOCK_DIM (4)
#define BUILD_DESTINATION_TABLE_TEST_R (2)
#define BUILD_DESTINATION_TABLE_TEST_BIT_GROUP (2)
#define BUILD_DESTINATION_TABLE_TEST_MIN 0
#define BUILD_DESTINATION_TABLE_TEST_MAX 10

void TestBuildDestinationTable()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test Build Destination Table --" << std::endl;
	#endif

	int count = BUILD_DESTINATION_TABLE_TEST_COUNT;
	int r = BUILD_DESTINATION_TABLE_TEST_R;
	int bit_group
		int block = BUILD_DESTINATION_TABLE_TEST_BLOCK_DIM;
	int grid = (count + block - 1) / block;
	int h_count = (1 << r);
	int h_total_count = h_count * grid;
	int block_sums_count = GetGPUPrefixSumBlockSumsCount(h_total_count, block);

	// TODO: compute sizes
	// TODO: allocate
	// TODO: cpu
	// TODO: print timinngs
	// TODO: gpu
	// TODO: print timings
	// TODO: check arrays
	// TODO: deallocate
}
#endif

/*
	a: input array
	l: local offsets
	g: global offsets
*/
__global__ void LSDRadixSortKernel(uint32_t* a, uint32_t* l, uint32_t* g, int count, int r, int bit_group)
{
	extern __shared__ uint32_t smem[];

	int tid = threadIdx.x;
	int bdim = blockDim.x;
	int bid = blockIdx.x;
	int idx = bid * bdim + tid;

	if (idx > count) return;

	int h_count = (1 << r);
	float cells_per_thread_ratio = (float)(h_count) / (float)(bdim);
	int cells_per_thread = cells_per_thread_ratio < 1.0f ? 1 : (int)(cells_per_thread_ratio + 0.5f);

	uint32_t* smem_a = smem;                      // elements
	uint32_t* smem_d = &smem[bdim];               // destination table
	uint32_t* smem_l = &smem[bdim * 2];           // local offsets
	uint32_t* smem_g = &smem[bdim * 2 + h_count]; // global offsets

	// load data into smem 
	smem_a[tid] = a[idx];
	for (int i = 0; i < cells_per_thread; i++)
	{
		int smem_h_i = tid * cells_per_thread + i;
		int global_h_i = bid * h_count + smem_h_i;
		if (smem_h_i < h_count)
		{
			smem_l[smem_h_i] = l[global_h_i];
			smem_g[smem_h_i] = g[global_h_i];
		}
	}
	__syncthreads();

	// sort smem_a
	SMEMLSDBinaryRadixSort(smem_a, tid, bdim, bit_group * r, r);
	// build destination table
	uint32_t val = smem_a[tid];
	int key = GET_R_BITS(val, r, bit_group);
	smem_d[tid] = (uint32_t)((int64_t)tid - (int64_t)(smem_l[key]) + (int64_t)(smem_g[key]));
	__syncthreads();

	// scatter elements using destination table
	a[smem_d[tid]] = smem_a[tid];
}

__global__ void BuildDestinationTableKernel(uint32_t* a, uint32_t* l, uint32_t* g, uint32_t* d, int count, int r, int bit_group)
{
	int tid = threadIdx.x;
	int bdim = blockDim.x;
	int bid = blockIdx.x;
	int idx = bid * bdim + tid;

	if (idx > count) return;

	uint32_t val = a[idx];
	int key = GET_R_BITS(val, r, bit_group);
	int h_count = (1 << r);
	d[idx] = (uint32_t)((int64_t)tid - (int64_t)(l[bid * h_count + key]) + (int64_t)(g[bid * h_count + key]));
}

__global__ void ScatterKernel(uint32_t* a, uint32_t* d, int count)
{
	int tid = threadIdx.x;
	int bdim = blockDim.x;
	int bid = blockIdx.x;
	int idx = bid * bdim + tid;

	if (idx > count) return;

	uint32_t value = a[idx];
	uint32_t destination = d[idx];
	__syncthreads();

	a[destination] = value;
}

//#define LSD_RADIX_SORT_DBG_PRINT
#define LSD_RADIX_SORT_VALIDATE

void GPULSDRadixSort(uint32_t* a, uint32_t* h, uint32_t* block_sums, uint32_t* d, int grid, int block, int block_sums_count, int count, int h_count, int r)
{
	hipStream_t s1 = MyCudaStreamCreate();
	hipStream_t s2 = MyCudaStreamCreate();

	#if defined(LSD_RADIX_SORT_DBG_PRINT) || defined(LSD_RADIX_SORT_VALIDATE)
	auto tmp_a = new uint32_t[count];
	auto tmp_d = new uint32_t[count];
	auto tmp_h = new uint32_t[h_count * grid];
	auto tmp_l = new uint32_t[h_count * grid];
	auto tmp_g = new uint32_t[h_count * grid];
	#endif

	#if defined(LSD_RADIX_SORT_VALIDATE)
	auto tmp_a0 = new uint32_t[count];
	auto tmp_a1 = new uint32_t[count];
	auto tmp_d0 = new uint32_t[count];
	auto tmp_h0 = new uint32_t[h_count * grid];
	auto tmp_h1 = new uint32_t[h_count];
	auto tmp_l0 = new uint32_t[h_count * grid];
	auto tmp_g0 = new uint32_t[h_count * grid];
	auto tmp_g1 = new uint32_t[h_count * grid];
	#endif

	#if defined(LSD_RADIX_SORT_DBG_PRINT) || defined(LSD_RADIX_SORT_VALIDATE)
	CUDA_CALL(hipMemcpy(tmp_a, a, count * sizeof(uint32_t), hipMemcpyDeviceToHost));
	memcpy(tmp_a0, tmp_a, count * sizeof(uint32_t));
	#endif
	#if defined(LSD_RADIX_SORT_DBG_PRINT)
	PrintArray('a', tmp_a, count);
	#endif

	int bit_groups = (sizeof(*a) * 8) / r;
	for (int bit_group = 0; bit_group < bit_groups; bit_group++)
	{
		// Build histogram
		{
			size_t smem = h_count * sizeof(uint32_t);
			BuildHistogramsKernel << <grid, block, smem >> > (a, h, count, r, bit_group);
			#if defined(LSD_RADIX_SORT_DBG_PRINT) || defined(LSD_RADIX_SORT_VALIDATE)
			CUDA_CALL(hipMemcpy(tmp_h, h, h_count * grid * sizeof(uint32_t), hipMemcpyDeviceToHost));
			#endif
			#if defined(LSD_RADIX_SORT_VALIDATE)
			memset(tmp_h0, 0, h_count * grid * sizeof(uint32_t));
			BuildHistogramsCPU(tmp_a, tmp_h0, count, r, bit_group, grid, block);
			CheckArrays(tmp_h0, tmp_h, h_count * grid);
			#endif
			#if defined(LSD_RADIX_SORT_DBG_PRINT)
			PrintArray('h', tmp_h, h_count * grid);
			#endif
		}

		// Build local and global offsets
		uint32_t* local_offsets = nullptr;
		uint32_t* global_offsets = nullptr;
		{
			int h_total_count = h_count * grid;
			size_t h_total_size = h_total_count * sizeof(uint32_t);
			local_offsets = h;
			global_offsets = &h[h_total_count];
			uint32_t* transposed_global_offsets = &h[h_total_count * 2];
			CUDA_CALL(hipMemcpy(global_offsets, local_offsets, h_total_size, hipMemcpyDeviceToDevice));

			// Build local offsets
			{
				int h_block = h_count;
				int h_grid = (h_total_count + h_block - 1) / h_block;
				size_t h_smem = h_block * sizeof(uint32_t);
				BlockPrefixSumKernel << <h_grid, h_block, h_smem, s1 >> > (local_offsets, nullptr);
				#if defined(LSD_RADIX_SORT_DBG_PRINT) || defined(LSD_RADIX_SORT_VALIDATE)
				CUDA_CALL(hipMemcpy(tmp_l, local_offsets, h_count * grid * sizeof(uint32_t), hipMemcpyDeviceToHost));
				#endif
				#if defined(LSD_RADIX_SORT_VALIDATE)
				memcpy(tmp_l0, tmp_h0, h_count * grid * sizeof(uint32_t));
				for (int g = 0; g < h_grid; g++)
				{
					PrefixSum(tmp_l0 + g * h_block, h_block);
				}
				CheckArrays(tmp_l0, tmp_l, h_count * grid);
				#endif
				#if defined(LSD_RADIX_SORT_DBG_PRINT)
				PrintArray('l', tmp_l, h_count * grid);
				#endif
			}

			// Build global offsets
			{
				int rows = grid;
				int cols = h_count;
				int block_dim = 32;
				{
					dim3 t_block(block_dim, block_dim);
					dim3 t_grid((cols + block_dim - 1) / block_dim, (rows + block_dim - 1) / block_dim);
					size_t t_smem = block_dim * block_dim * sizeof(uint32_t);
					TransposeSMEMKernel << <t_grid, t_block, t_smem, s2 >> > (global_offsets, transposed_global_offsets, rows, cols);
				}
				GPUPrefixSum(transposed_global_offsets, h_total_count, block, block_sums, s2);
				{
					dim3 t_block(block_dim, block_dim);
					dim3 t_grid((rows + block_dim - 1) / block_dim, (cols + block_dim - 1) / block_dim);
					size_t t_smem = block_dim * block_dim * sizeof(uint32_t);
					TransposeSMEMKernel << <t_grid, t_block, t_smem, s2 >> > (transposed_global_offsets, global_offsets, cols, rows);
				}
				#if defined(LSD_RADIX_SORT_DBG_PRINT) || defined(LSD_RADIX_SORT_VALIDATE)
				CUDA_CALL(hipMemcpy(tmp_g, global_offsets, h_count * grid * sizeof(uint32_t), hipMemcpyDeviceToHost));
				#endif
				#if defined(LSD_RADIX_SORT_VALIDATE)
				memcpy(tmp_g0, tmp_h0, h_count * grid * sizeof(uint32_t));
				Transpose(tmp_g0, tmp_g1, rows, cols);
				PrefixSum(tmp_g1, h_count * grid);
				Transpose(tmp_g1, tmp_g0, cols, rows);
				CheckArrays(tmp_g0, tmp_g, h_count * grid);
				#endif
				#if defined(LSD_RADIX_SORT_DBG_PRINT)
				PrintArray('g', tmp_g, h_count * grid);
				#endif
			}
		}

		// Sort
		{
			// Block local sort
			size_t smem = block * sizeof(uint32_t);
			int first_bit = bit_group * r;
			int bit_count = r;
			LSDBinaryRadixSortKernel << <grid, block, smem >> > (a, first_bit, bit_count);
			CUDA_CALL(hipDeviceSynchronize());
			#if defined(LSD_RADIX_SORT_DBG_PRINT) || defined(LSD_RADIX_SORT_VALIDATE)
			CUDA_CALL(hipMemcpy(tmp_a, a, count * sizeof(uint32_t), hipMemcpyDeviceToHost));
			#endif
			#if defined(LSD_RADIX_SORT_VALIDATE)
			for (int g = 0; g < grid; g++)
			{
				LSDRadixSortPass(tmp_a0 + g * block, tmp_a1 + g * block, block, tmp_h1, r, bit_group);
			}
			CheckArrays(tmp_a1, tmp_a, count);
			#endif
			#if defined(LSD_RADIX_SORT_DBG_PRINT)
			PrintArray('a', tmp_a, count);
			#endif

			// Build destination table
			BuildDestinationTableKernel << <grid, block >> > (a, local_offsets, global_offsets, d, count, r, bit_group);
			#if defined(LSD_RADIX_SORT_DBG_PRINT) || defined(LSD_RADIX_SORT_VALIDATE)
			CUDA_CALL(hipMemcpy(tmp_d, d, count * sizeof(uint32_t), hipMemcpyDeviceToHost));
			#endif
			#if defined(LSD_RADIX_SORT_VALIDATE)
			for (int i = 0; i < count; i++)
			{
				int bid = i / block;
				int tid = i % block;

				uint32_t val = tmp_a1[i];
				int key = GET_R_BITS(val, r, bit_group);
				tmp_d0[i] = (uint32_t)((int64_t)tid - (int64_t)(tmp_l[bid * h_count + key]) + (int64_t)(tmp_g[bid * h_count + key]));
			}
			CheckArrays(tmp_d0, tmp_d, count);
			#endif
			#if defined(LSD_RADIX_SORT_DBG_PRINT)
			PrintArray('d', tmp_d, count);
			#endif

			// Scatter
			ScatterKernel << <grid, block >> > (a, d, count);
			#if defined(LSD_RADIX_SORT_DBG_PRINT) || defined(LSD_RADIX_SORT_VALIDATE)
			CUDA_CALL(hipMemcpy(tmp_a, a, count * sizeof(uint32_t), hipMemcpyDeviceToHost));
			#endif
			#if defined(LSD_RADIX_SORT_VALIDATE)
			for (int i = 0; i < count; i++)
			{
				tmp_a0[tmp_d0[i]] = tmp_a1[i];
			}
			CheckArrays(tmp_a0, tmp_a, count);
			#endif
			#if defined(LSD_RADIX_SORT_DBG_PRINT)
			PrintArray('a', tmp_a, count);
			#endif
		}
	}

	CUDA_CALL(hipStreamDestroy(s2));
	CUDA_CALL(hipStreamDestroy(s1));
}

#define GPU_LSD_SORT_TEST_COUNT (1024 * 2)
#define GPU_LSD_SORT_TEST_BLOCK_DIM (2)
#define GPU_LSD_SORT_TEST_R (2)
#define GPU_LSD_SORT_TEST_MIN 0
#define GPU_LSD_SORT_TEST_MAX 10

void TestGPULSDRadixSort()
{
	// print header
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test GPU LSD Radix Sort --" << std::endl;
	#endif

	int r = GPU_LSD_SORT_TEST_R;
	int count = GPU_LSD_SORT_TEST_COUNT;
	int block = GPU_LSD_SORT_TEST_BLOCK_DIM;
	int grid = (count + block - 1) / block;
	int h_count = (1 << r);
	int h_total_count = h_count * grid;
	int block_sums_count = GetGPUPrefixSumBlockSumsCount(h_total_count, block);

	// get sizes
	size_t size = count * sizeof(uint32_t);
	size_t h_total_size = h_total_count * sizeof(uint32_t);
	size_t h_triple_total_size = 3 * h_total_size;
	size_t block_sums_size = block_sums_count * sizeof(uint32_t);
	size_t total_size = h_triple_total_size + block_sums_size;

	// print sizes and config
	#ifdef PRINT_TIMINGS
	std::cout << "Elements: " << (double)(size) / (1024.0 * 1024.0 * 1024.0) << " GB" << std::endl;
	std::cout << "Histograms: " << (double)(h_triple_total_size) / (1024.0 * 1024.0 * 1024.0) << " GB" << std::endl;
	std::cout << "Block Sums: " << (double)(block_sums_size) / (1024.0 * 1024.0 * 1024.0) << " GB" << std::endl;
	std::cout << "Block Size: " << block << std::endl;
	std::cout << "R: " << r << std::endl;
	#endif

	/*
	if (total_size > size)
	{
		#ifdef PRINT_TIMINGS
		std::cout << "SKIP: auxiliary data is more than input" << std::endl;
		#endif

		#ifdef BENCHMARK_GPU_LSD_RADIX_SORT
		return;
		#else
		MYCRASH();
		#endif
	}
	*/

	if (r > 10)
	{
		#ifdef PRINT_TIMINGS
		std::cout << "SKIP: R is too big for block local prefix sum" << std::endl;
		#endif

		#ifdef BENCHMARK_GPU_LSD_RADIX_SORT
		return;
		#else
		MYCRASH();
		#endif
	}

	// allocate
	uint32_t* a = (uint32_t*)calloc(1, size);
	uint32_t* b = (uint32_t*)calloc(1, size);
	uint32_t* h = (uint32_t*)calloc(h_count, sizeof(*h));
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_d = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_h = (uint32_t*)MyCudaMalloc(h_triple_total_size);
	uint32_t* d_block_sums = (uint32_t*)MyCudaMalloc(block_sums_size);

	// populate input
	RNG rng = RNG(0, GPU_LSD_SORT_TEST_MIN, GPU_LSD_SORT_TEST_MAX);
	for (int i = 0; i < count; i++) a[i] = rng.Get();
	memcpy(h_a, a, size);

	// cpu
	float cpu_ms = 0.0f;
	{
		int64_t start = GetTimestamp();
		LSDRadixSort(a, b, count, h, r);
		int64_t end = GetTimestamp();
		cpu_ms = GetElapsedMS(start, end);
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "CPU " << cpu_ms << " ms" << std::endl;
	#endif

	// gpu
	float gpu_ms = 0.0f;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		GPULSDRadixSort(d_a, d_h, d_block_sums, d_d, grid, block, block_sums_count, count, h_count, r);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_a, d_a, size, hipMemcpyDeviceToHost));
		gpu_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "GPU " << gpu_ms << " ms - Speedup: x" << cpu_ms / gpu_ms << std::endl;
	#endif

	// check arrays
	CheckArrays(h_a, b, count);

	// deallocate
	CUDA_CALL(hipFree(d_block_sums));
	CUDA_CALL(hipFree(d_h));
	CUDA_CALL(hipFree(d_d));
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_a));
	free(h);
	free(b);
	free(a);
}

void BenchmarkGPUPrefixSum()
{
	int count[] =
	{
		1024 * 1024 * 64,
		1024 * 1024 * 128,
		1024 * 1024 * 256,
		1024 * 1024 * 512,
		1024 * 1024 * 1024,
	};

	int threads_per_block[] =
	{
		64,
		128,
		256,
		512,
		1024,
	};

	for (int i = 0; i < MYARRAYCOUNT(threads_per_block); i++)
	{
		for (int j = 0; j < MYARRAYCOUNT(count); j++)
		{
			TestGPUPrefixSum(count[i], threads_per_block[j], 0, 10);
		}
	}
}

#define PREFIX_SUM_TEST_ELEMS_COUNT (1024 * 1024)
#define PREFIX_SUM_TEST_ELEMS_THREADS_PER_BLOCK (128)
#define PREFIX_SUM_TEST_ELEMS_MIN 0
#define PREFIX_SUM_TEST_ELEMS_MAX 10

#define BUILD_HISTOGRAM_TEST_ELEMS_COUNT (1024 * 1024)
#define BUILD_HISTOGRAM_TEST_BLOCK_DIM (256)
#define BUILD_HISTOGRAM_TEST_MIN 0
#define BUILD_HISTOGRAM_TEST_MAX 10
#define BUILD_HISTOGRAM_TEST_BIT_GROUP 0
#define BUILD_HISTOGRAM_TEST_R 4

int main()
{
	CheckForHostLeaks();

	#ifdef BENCHMARK_BUILD_HISTOGRAMS
	BenchmarkBuildHistogram();
	#else
	TestSequentialLSDRadixSort();
	TestBlockPrefixSumKernel();
	TestGPUPrefixSum(PREFIX_SUM_TEST_ELEMS_COUNT, PREFIX_SUM_TEST_ELEMS_THREADS_PER_BLOCK, PREFIX_SUM_TEST_ELEMS_MIN, PREFIX_SUM_TEST_ELEMS_MAX);
	TestLSDBinaryRadixSort();
	TestTranspose();
	TestBuildHistogram(BUILD_HISTOGRAM_TEST_ELEMS_COUNT, BUILD_HISTOGRAM_TEST_BLOCK_DIM, BUILD_HISTOGRAM_TEST_R, BUILD_HISTOGRAM_TEST_BIT_GROUP, BUILD_HISTOGRAM_TEST_MIN, BUILD_HISTOGRAM_TEST_MAX);
	//TestBuildDestinationTable();
	TestGPULSDRadixSort();
	#endif

	return 0;
}
