#include "hip/hip_runtime.h"
#include <iostream>
#include <stdint.h>

/*
	TODO: watch this https://www.youtube.com/watch?v=fsC3QeZHM1U
*/

// Windows only
#define _CRTDBG_MAP_ALLOC
#include <stdlib.h>
#include <crtdbg.h>
#include <intrin.h>

#include "Utils.h"
#include "CudaUtils.h"

/*
* Extract the i-th group of r bits from n
*/
#define GET_R_BITS(n, r, i) (((1 << r) - 1) & (n >> (i * r)))

/*
* in:    input array (will be modified)
* out:   output array
* count: number of elements to sort
* r:     number of bits to consider as keys (any factor of 32 apart from itself)
*/
void LSDRadixSort(uint32_t* in, uint32_t* out, int count, uint32_t* histogram, int r)
{
	int iterations = (sizeof(*in) * 8) / r;
	for (int i = 0; i < iterations; i++)
	{
		memset(histogram, 0, sizeof(*histogram) * (1 << r));

		// build histogram
		for (int j = 0; j < count; j++)
		{
			uint32_t val = in[j];
			int key = GET_R_BITS(val, r, i);
			histogram[key]++;
		}

		// prefix sum of histogram
		for (int j = 1; j < (1 << r); j++)
		{
			histogram[j] += histogram[j - 1];
		}

		// permute input elements into output
		for (int j = count - 1; j >= 0; j--)
		{
			uint32_t val = in[j];
			int key = GET_R_BITS(val, r, i);
			histogram[key]--;
			out[histogram[key]] = val;
		}

		// copy output array into input array
		memcpy(in, out, sizeof(*in) * count);
	}
}

void PrefixSum(uint32_t* a, int count)
{
	for (int i = 1; i < count; i++)
	{
		a[i] += a[i - 1];
	}
	for (int i = count - 1; i >= 1; i--)
	{
		a[i] = a[i - 1];
	}
	a[0] = 0;
}

__device__ void SMEMUpSweep(uint32_t* smem, int bdim, int tid)
{
	for (int d = 0; (1 << d) < bdim; d++)
	{
		int offset = (1 << (d + 1));
		int bias = offset - 1;
		int shift = (1 << d);

		if (tid < (bdim >> (d + 1)))
		{
			int index = bias + tid * offset;
			int left = index - shift;
			smem[index] += smem[left];
		}
		__syncthreads();
	}
}

__device__ void SMEMDownSweep(uint32_t* smem, int bdim, int tid)
{
	for (int d = 0; (1 << d) < bdim; d++)
	{
		int offset = (bdim >> d);
		int bias = offset - 1;
		int shift = (bdim >> (d + 1));

		if (tid < (1 << d))
		{
			int index = bias + tid * offset;
			int left = index - shift;
			int l = smem[index];
			int r = smem[index] + smem[left];
			smem[left] = l;
			smem[index] = r;
		}
		__syncthreads();
	}
}

__global__ void BlockPrefixSumKernel(uint32_t* a, uint32_t* block_sums)
{
	extern __shared__ uint32_t smem[];

	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int bdim = blockDim.x;
	int i = bid * bdim + tid;

	// load array into smem
	smem[tid] = a[i];
	__syncthreads();

	SMEMUpSweep(smem, bdim, tid);
	if (tid == 0)
	{
		block_sums[bid] = smem[bdim - 1];
		smem[bdim - 1] = 0;
	}
	__syncthreads();
	SMEMDownSweep(smem, bdim, tid);

	// write smem into array
	a[i] = smem[tid];
}

#define PRINT_TIMINGS

#define SEQ_RADIX_SORT_TEST_ELEMS_COUNT 16
#define SEQ_RADIX_SORT_TEST_ELEMS_MIN 0
#define SEQ_RADIX_SORT_TEST_ELEMS_MAX UINT32_MAX
#define SEQ_RADIX_SORT_TEST_ELEMS_R 4

void TestSequentialLSDRadixSort()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test sequential LSD radix sort --" << std::endl;
	#endif

	RNG rng = RNG(0, SEQ_RADIX_SORT_TEST_ELEMS_MIN, SEQ_RADIX_SORT_TEST_ELEMS_MAX);

	int count = SEQ_RADIX_SORT_TEST_ELEMS_COUNT;
	uint32_t* a = (uint32_t*)(calloc(count, sizeof(*a)));
	uint32_t* b = (uint32_t*)(calloc(count, sizeof(*b)));
	uint32_t* c = (uint32_t*)(calloc(count, sizeof(*c)));
	uint32_t* h = (uint32_t*)(calloc(1 << SEQ_RADIX_SORT_TEST_ELEMS_R, sizeof(*h)));

	// populate a and c
	for (int i = 0; i < count; i++)
	{
		int elem = rng.Get();
		a[i] = elem;
		c[i] = elem;
	}

	// sort a writing result in b using LSD radix sort
	float lsd_radix_sort_ms = 0;
	{
		int64_t start = GetTimestamp();
		LSDRadixSort(a, b, count, h, SEQ_RADIX_SORT_TEST_ELEMS_R);
		int64_t end = GetTimestamp();
		lsd_radix_sort_ms = GetElapsedMS(start, end);
	}

	// sort c using standard library sort
	float std_sort_ms = 0;
	{
		int64_t start = GetTimestamp();
		std::sort(c, c + count);
		int64_t end = GetTimestamp();
		std_sort_ms = GetElapsedMS(start, end);
	}

	#ifdef PRINT_ARRAY
	PrintArray('b', b, count);
	PrintArray('c', c, count);
	#endif

	#ifdef PRINT_TIMINGS
	std::cout << "Sequential LSD Radix Sort: " << lsd_radix_sort_ms << " ms" << std::endl;
	std::cout << "Sequential STD Sort: " << std_sort_ms << " ms" << std::endl;
	std::cout << "Speedup: x" << std_sort_ms / lsd_radix_sort_ms << std::endl;
	#endif

	CheckArrays(b, c, count);

	free(h);
	free(c);
	free(b);
	free(a);
}

#define BLOCK_PREFIX_SUM_TEST_ELEMS_COUNT 1024
#define BLOCK_PREFIX_SUM_TEST_ELEMS_MIN 0
#define BLOCK_PREFIX_SUM_TEST_ELEMS_MAX 10

void TestBlockPrefixSumKernel()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test block exclusive prefix sum --" << std::endl;
	#endif

	RNG rng = RNG(0, BLOCK_PREFIX_SUM_TEST_ELEMS_MIN, BLOCK_PREFIX_SUM_TEST_ELEMS_MAX);

	int count = BLOCK_PREFIX_SUM_TEST_ELEMS_COUNT;
	size_t size = count * sizeof(uint32_t);
	size_t blocks_size = 1 * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_block_sums = (uint32_t*)MyCudaMalloc(blocks_size);

	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	float parallel_ms = 0;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		BlockPrefixSumKernel << <1, count, size >> > (d_a, d_block_sums);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost));
		parallel_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	float sequential_ms = 0;
	{
		int64_t start = GetTimestamp();
		PrefixSum(h_a, count);
		int64_t end = GetTimestamp();
		sequential_ms = GetElapsedMS(start, end);
	}

	#ifdef PRINT_ARRAY
	PrintArray('a', h_a, count);
	PrintArray('b', h_b, count);
	#endif

	#ifdef PRINT_TIMINGS
	std::cout << "Prefix Sum Sequential: " << sequential_ms << " ms" << std::endl;
	std::cout << "Prefix Sum Block: " << parallel_ms << " ms" << std::endl;
	std::cout << "Speedup: x" << sequential_ms / parallel_ms << std::endl;
	#endif

	CheckArrays(h_a, h_b, count);

	CUDA_CALL(hipFree(d_block_sums));
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_b));
	CUDA_CALL(hipHostFree(h_a));
}

int GetGPUPrefixSumBlockSumsCount(int count, int threads_per_block)
{
	MYASSERT(count % threads_per_block == 0);
	int total_block_sums_count = 0;
	while (count > threads_per_block)
	{
		MYASSERT(count % threads_per_block == 0);
		int block_sums_count = count / threads_per_block;
		total_block_sums_count += block_sums_count;
		count = block_sums_count;
	}
	return total_block_sums_count + 1;
}

__global__ void AddBlockSumsKernel(uint32_t* a, uint32_t* block_sums)
{
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int i = blockDim.x * bid + tid;
	a[i] += block_sums[bid];
}

void GPUPrefixSum(uint32_t* d_a, int count, int threads_per_block, uint32_t* d_block_sums)
{
	if (count <= threads_per_block)
	{
		int smem = count * sizeof(uint32_t);
		BlockPrefixSumKernel << <1, count, smem >> > (d_a, d_block_sums);
	}
	else
	{
		int smem = threads_per_block * sizeof(uint32_t);
		int blocks = count / threads_per_block;
		BlockPrefixSumKernel << <blocks, threads_per_block, smem >> > (d_a, d_block_sums);
		GPUPrefixSum(d_block_sums, blocks, threads_per_block, &d_block_sums[blocks]);
		// Skip the first block, since the exclusive block sum will have 0
		AddBlockSumsKernel << <blocks - 1, threads_per_block >> > (&d_a[threads_per_block], &d_block_sums[1]);
	}
}

void TestGPUPrefixSum(int count, int threads_per_block, int min, int max)
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test exclusive prefix sum --" << std::endl;
	#endif

	RNG rng = RNG(0, min, max);

	// allocate
	int block_sums_count = GetGPUPrefixSumBlockSumsCount(count, threads_per_block);
	size_t size = count * sizeof(uint32_t);
	size_t block_sums_size = block_sums_count * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_block_sums = (uint32_t*)MyCudaMalloc(block_sums_size);

	// populate input
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	// parallel prefix sum
	float parallel_ms = 0;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		GPUPrefixSum(d_a, count, threads_per_block, d_block_sums);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost));
		parallel_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// sequential prefix sum
	float sequential_ms = 0;
	{
		int64_t start = GetTimestamp();
		PrefixSum(h_a, count);
		int64_t end = GetTimestamp();
		sequential_ms = GetElapsedMS(start, end);
	}

	// print arrays
	#ifdef PRINT_ARRAY
	PrintArray('a', h_a, count);
	PrintArray('b', h_b, count);
	#endif

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "Prefix sum of " << (double)(size) / (1024.0 * 1024.0 * 1024.0) << " GB of data" << std::endl;
	std::cout << "Threads per block: " << threads_per_block << std::endl;
	std::cout << "Prefix Sum Sequential: " << sequential_ms << " ms" << std::endl;
	std::cout << "GPU Prefix Sum: " << parallel_ms << " ms" << std::endl;
	std::cout << "Speedup: x" << sequential_ms / parallel_ms << std::endl;
	#endif

	// check arrays
	CheckArrays(h_a, h_b, count);

	// deallocate
	CUDA_CALL(hipFree(d_block_sums));
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_b));
	CUDA_CALL(hipHostFree(h_a));
}

__device__ void SMEMLSDBinaryRadixSort(uint32_t* a, int tid, int bdim)
{
	// 32 passes for 32 bits numbers
	for (int i = 0; i < 32; i++)
	{
		uint32_t val = a[tid];
		uint32_t bit = GET_R_BITS(val, 1, i);
		// invert bit and write it to a
		a[tid] = bit ? 0 : 1;
		__syncthreads();

		// prefix sum of inverted bits
		SMEMUpSweep(a, bdim, tid);
		uint32_t total_falses = a[bdim - 1];
		if (tid == 0) a[bdim - 1] = 0;
		__syncthreads();
		SMEMDownSweep(a, bdim, tid);

		// now a holds the destination for false keys
		// destination for true key
		uint32_t t = (uint32_t)tid - a[tid] + total_falses;
		// destination for val
		uint32_t d = bit ? t : a[tid];
		a[d] = val;
	}
	__syncthreads();
}

__global__ void LSDBinaryRadixSortKernel(uint32_t* a)
{
	extern __shared__ uint32_t smem[];

	int tid = threadIdx.x;

	// load array in smem
	smem[tid] = a[tid];
	__syncthreads();

	SMEMLSDBinaryRadixSort(smem, tid, blockDim.x);

	// write smem into array
	a[tid] = smem[tid];
}

#define LSD_BINARY_RADIX_SORT_TEST_ELEMS_COUNT (1024)
#define LSD_BINARY_RADIX_SORT_TEST_ELEMS_MIN 0
#define LSD_BINARY_RADIX_SORT_TEST_ELEMS_MAX UINT32_MAX

void TestLSDBinaryRadixSort()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test LSD Binary Radix Sort --" << std::endl;
	#endif

	RNG rng = RNG(0, LSD_BINARY_RADIX_SORT_TEST_ELEMS_MIN, LSD_BINARY_RADIX_SORT_TEST_ELEMS_MAX);

	// allocate
	int count = LSD_BINARY_RADIX_SORT_TEST_ELEMS_COUNT;
	size_t size = count * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);

	// populate input
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	// parallel lsd binary radix sort
	float parallel_lsd_binary_radix_sort_ms = 0;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		LSDBinaryRadixSortKernel << <1, count, size >> > (d_a);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_b, d_a, size, hipMemcpyDeviceToHost));
		parallel_lsd_binary_radix_sort_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// sequential sort
	float std_sort_ms = 0;
	{
		int64_t start = GetTimestamp();
		std::sort(h_a, h_a + count);
		int64_t end = GetTimestamp();
		std_sort_ms = GetElapsedMS(start, end);
	}

	// print arrays
	#ifdef PRINT_ARRAY
	PrintArray('a', h_a, count);
	PrintArray('b', h_b, count);
	#endif

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "STD Sort: " << std_sort_ms << " ms" << std::endl;
	std::cout << "Parallel LSD Binary Radix Sort: " << parallel_lsd_binary_radix_sort_ms << " ms" << std::endl;
	#endif

	// check arrays
	CheckArrays(h_a, h_b, count);

	// deallocate
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_b));
	CUDA_CALL(hipHostFree(h_a));
}

/*
	a: m x n matrix
	b: n x m matrix
*/
void Transpose(uint32_t* a, uint32_t* b, int m, int n)
{
	for (int row = 0; row < m; row++)
	{
		for (int col = 0; col < n; col++)
		{
			int a_i = row * n + col;
			int b_i = col * m + row;
			b[b_i] = a[a_i];
		}
	}
}

/*
	a : m x n matrix
	b : n x m matrix
*/
__global__ void TransposeNaiveKernel(uint32_t* a, uint32_t* b, int m, int n)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= m || j >= n) return;
	b[j * m + i] = a[i * n + j];
}

/*
	a : m x n matrix
	b : n x m matrix
*/
__global__ void TransposeSMEMKernel(uint32_t* a, uint32_t* b, int m, int n)
{
	extern __shared__ uint32_t smem[];

	int t_i = threadIdx.y;
	int t_j = threadIdx.x;
	int t_cols = blockDim.x;
	int t_rows = blockDim.y;
	int a_i = blockIdx.y * blockDim.y + t_i;
	int a_j = blockIdx.x * blockDim.x + t_j;
	int b_i = blockIdx.x * blockDim.x + t_i;
	int b_j = blockIdx.y * blockDim.y + t_j;

	// copy matrix block into smem
	if (a_i < m && a_j < n)
	{
		smem[t_i * t_cols + t_j] = a[a_i * n + a_j];
	}
	__syncthreads();

	// write transposed smem block
	if (b_i < n && b_j < m)
	{
		b[b_i * m + b_j] = smem[t_j * t_rows + t_i];
	}
}

#define TRANSPOSE_TEST_M (1024)
#define TRANSPOSE_TEST_N (1024)
#define TRANSPOSE_TEST_BLOCK_DIM 32
#define TRANSPOSE_TEST_MIN_ELEM 0
#define TRANSPOSE_TEST_MAX_ELEM 9

void TestTranspose()
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test Transpose --" << std::endl;
	#endif

	RNG rng = RNG(0, TRANSPOSE_TEST_MIN_ELEM, TRANSPOSE_TEST_MAX_ELEM);

	// allocate
	int m = TRANSPOSE_TEST_M;
	int n = TRANSPOSE_TEST_N;
	int count = m * n;
	int block_dim = TRANSPOSE_TEST_BLOCK_DIM;
	size_t size = count * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_c = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_b = (uint32_t*)MyCudaMalloc(size);

	#ifdef PRINT_TIMINGS
	std::cout << "Transpose of " << (double)(size) / (1024.0 * 1024.0 * 1024.0) << " GB of data" << std::endl;
	#endif

	// populate input
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	// sequential transpose
	float sequential_ms = 0.0f;
	{
		int64_t start = GetTimestamp();
		Transpose(h_a, h_b, m, n);
		int64_t end = GetTimestamp();
		sequential_ms = GetElapsedMS(start, end);
	}

	float gpu_naive_ms = 0.0f;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		dim3 block(block_dim, block_dim);
		dim3 grid((n + block_dim - 1) / block_dim, (m + block_dim - 1) / block_dim);
		TransposeNaiveKernel << <grid, block >> > (d_a, d_b, m, n);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_c, d_b, size, hipMemcpyDeviceToHost));
		gpu_naive_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "Sqeuential Transpose: " << sequential_ms << " ms" << std::endl;
	std::cout << "GPU Naive Transpose: " << gpu_naive_ms << " ms - Speedup: x" << sequential_ms / gpu_naive_ms << std::endl;
	#endif

	// check matrices
	CheckArrays(h_b, h_c, count);

	float gpu_smem_ms = 0.0f;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		dim3 block(block_dim, block_dim);
		dim3 grid((n + block_dim - 1) / block_dim, (m + block_dim - 1) / block_dim);
		size_t smem = block_dim * block_dim * sizeof(uint32_t);
		TransposeSMEMKernel << <grid, block, smem >> > (d_a, d_b, m, n);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_c, d_b, size, hipMemcpyDeviceToHost));
		gpu_smem_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "GPU SMEM Transpose: " << gpu_smem_ms << " ms - Speedup: x" << sequential_ms / gpu_smem_ms << std::endl;
	#endif

	// check matrices
	CheckArrays(h_b, h_c, count);

	// deallocate
	hipFree(d_b);
	hipFree(d_a);
	hipHostFree(h_c);
	hipHostFree(h_b);
	hipHostFree(h_a);
}

/*
	bdim: 64   r: 1  lct: 2     ratio: 0.03125
	bdim: 64   r: 2  lct: 4     ratio: 0.0625
	bdim: 64   r: 4  lct: 16    ratio: 0.25
	bdim: 64   r: 8  lct: 256   ratio: 4
	bdim: 64   r: 16 lct: 65536 ratio: 1024

	bdim: 128  r: 1  lct: 2	    ratio: 0.015625
	bdim: 128  r: 2  lct: 4	    ratio: 0.03125
	bdim: 128  r: 4  lct: 16    ratio: 0.125
	bdim: 128  r: 8  lct: 256   ratio: 2
	bdim: 128  r: 16 lct: 65536 ratio: 512

	bdim: 256  r: 1  lct: 2	    ratio: 0.0078125
	bdim: 256  r: 2  lct: 4	    ratio: 0.015625
	bdim: 256  r: 4  lct: 16    ratio: 0.0625
	bdim: 256  r: 8  lct: 256   ratio: 1
	bdim: 256  r: 16 lct: 65536 ratio: 256

	bdim: 512  r: 1  lct: 2     ratio: 0.00390625
	bdim: 512  r: 2  lct: 4     ratio: 0.0078125
	bdim: 512  r: 4  lct: 16    ratio: 0.03125
	bdim: 512  r: 8  lct: 256   ratio: 0.5
	bdim: 512  r: 16 lct: 65536 ratio: 128

	bdim: 1024 r: 1  lct: 2     ratio: 001953125
	bdim: 1024 r: 2  lct: 4     ratio: 0.00390625
	bdim: 1024 r: 4  lct: 16    ratio: 0.015625
	bdim: 1024 r: 8  lct: 256   ratio: 0.25
	bdim: 1024 r: 16 lct: 65536 ratio: 64
*/

/*
	grid:  how many histograms
	block: how many array elements per histogram
*/
void BuildHistogramsCPU(uint32_t* a, uint32_t* h, int count, int r, int bit_group, int grid, int block)
{
	int h_count = (1 << r);

	for (int g = 0; g < grid; g++)
	{
		for (int b = 0; b < block; b++)
		{
			int a_i = g * block + b;
			uint32_t val = a[a_i];
			int key = GET_R_BITS(val, r, bit_group);
			int h_i = g * h_count + key;
			h[h_i] += 1;
		}
	}
}

__global__ void BuildHistogramsKernel(uint32_t* a, uint32_t* h, int count, int r, int bit_group)
{
	extern __shared__ uint32_t smem[];

	int bdim = blockDim.x;
	int tid = threadIdx.x;
	int bid = blockIdx.x;
	int idx = bid * bdim + tid;

	// zero initialize smem
	int lct_count = 1 << r;
	float cells_per_thread_ratio = (float)(lct_count) / (float)(bdim);
	int cells_per_thread = cells_per_thread_ratio < 1.0f ? 1 : (int)(cells_per_thread_ratio + 0.5f);
	for (int i = 0; i < cells_per_thread; i++)
	{
		int smem_i = tid * cells_per_thread + i;
		if (smem_i < lct_count)
		{
			smem[smem_i] = 0;
		}
	}
	__syncthreads();

	// build histogram in smem
	if (idx < count)
	{
		uint32_t val = a[idx];
		int key = GET_R_BITS(val, r, bit_group);
		atomicInc(&smem[key], UINT32_MAX);
	}
	__syncthreads();

	// write histogram to global memory
	for (int i = 0; i < cells_per_thread; i++)
	{
		int smem_i = tid * cells_per_thread + i;
		int h_i = bid * lct_count + smem_i;
		if (smem_i < lct_count)
		{
			h[h_i] = smem[smem_i];
		}
	}
}

void TestBuildHistogram(int count, int block, int r, int bit_group, int min, int max)
{
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test Build Histogram --" << std::endl;
	#endif

	int grid = (count + block - 1) / block;
	size_t h_count = (1 << r);
	size_t h_total_count = h_count * grid;

	// get sizes
	size_t size = count * sizeof(uint32_t);
	size_t h_total_size = h_total_count * sizeof(uint32_t);

	#ifdef PRINT_TIMINGS
	std::cout << "Elements: " << (double)(size) / (1024.0 * 1024.0 * 1024.0) << " GB" << std::endl;
	std::cout << "Histograms: " << (double)(h_total_size) / (1024.0 * 1024.0 * 1024.0) << " GB" << std::endl;
	std::cout << "Block Size: " << block << std::endl;
	std::cout << "R: " << r << std::endl;
	std::cout << "Bit Group: " << bit_group << std::endl;
	#endif

	if (h_total_size > size)
	{
		#ifdef PRINT_TIMINGS
		std::cout << "SKIP: histogram is bigger than input" << std::endl;
		#endif
		return;
	}

	// allocate
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_h1 = (uint32_t*)MyCudaHostAlloc(h_total_size);
	uint32_t* h_h2 = (uint32_t*)MyCudaHostAlloc(h_total_size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_h = (uint32_t*)MyCudaMalloc(h_total_size);

	// populate input
	RNG rng = RNG(0, min, max);
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	// cpu implementation
	float cpu_ms = 0.0f;
	{
		int64_t start = GetTimestamp();
		BuildHistogramsCPU(h_a, h_h1, count, r, bit_group, grid, block);
		int64_t end = GetTimestamp();
		cpu_ms = GetElapsedMS(start, end);
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "CPU " << cpu_ms << " ms" << std::endl;
	#endif

	// gpu implementation
	float gpu_ms = 0.0f;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		size_t smem = h_count * sizeof(uint32_t);
		BuildHistogramsKernel << <grid, block, smem >> > (d_a, d_h, count, r, bit_group);
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_h2, d_h, h_total_size, hipMemcpyDeviceToHost));
		gpu_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "GPU " << gpu_ms << " ms - Speedup: x" << cpu_ms / gpu_ms << std::endl;
	#endif

	// check arrays
	CheckArrays(h_h1, h_h2, h_total_count);

	// deallocate
	CUDA_CALL(hipFree(d_h));
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_h2));
	CUDA_CALL(hipHostFree(h_h1));
	CUDA_CALL(hipHostFree(h_a));
}

void BenchmarkBuildHistogram()
{
	int counts[] =
	{
		1024 * 1024 * 64,
		1024 * 1024 * 128,
		1024 * 1024 * 256,
		1024 * 1024 * 512,
		1024 * 1024 * 1024,
	};

	int blocks[] =
	{
		64,
		128,
		256,
		512,
		1024,
	};

	int rs[] =
	{
		1,
		2,
		4,
		8,
		16,
	};

	for (int c_i = 0; c_i < MYARRAYCOUNT(counts); c_i++)
	{
		for (int b_i = 0; b_i < MYARRAYCOUNT(blocks); b_i++)
		{
			for (int r_i = 0; r_i < MYARRAYCOUNT(rs); r_i++)
			{
				RNG rng = RNG(0, 0, (32 / rs[r_i]));
				TestBuildHistogram(counts[c_i], blocks[b_i], rs[r_i], rng.Get(), 0, UINT32_MAX);
			}
		}
	}
}

#define LSD_RADIX_SORT_TEST_ELEMS_COUNT (1024 * 2)
#define LSD_RADIX_SORT_TEST_BLOCK_DIM (1024)
#define LSD_RADIX_SORT_TEST_MIN 0
#define LSD_RADIX_SORT_TEST_MAX UINT32_MAX
#define LSD_RADIX_SORT_TEST_R 4

void TestLSDRadixSort()
{
	#if 0
	#ifdef PRINT_TIMINGS
	std::cout << "-- Test LSD Radix Sort --" << std::endl;
	#endif

	RNG rng = RNG(0, LSD_RADIX_SORT_TEST_MIN, LSD_RADIX_SORT_TEST_MAX);

	// allocate
	int block_dim = LSD_RADIX_SORT_TEST_BLOCK_DIM;
	int r = LSD_RADIX_SORT_TEST_R;
	int count = LSD_RADIX_SORT_TEST_ELEMS_COUNT;
	dim3 block(block_dim);
	dim3 grid((count + block_dim - 1) / block_dim);
	int histogram_count = (1 << r);
	size_t histogram_size = histogram_count * grid.x * sizeof(uint32_t);
	size_t size = count * sizeof(uint32_t);
	uint32_t* h_a = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_b = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* h_c = (uint32_t*)MyCudaHostAlloc(size);
	uint32_t* d_a = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_b = (uint32_t*)MyCudaMalloc(size);
	uint32_t* d_histogram = (uint32_t*)MyCudaMalloc(histogram_size);
	uint32_t* histogram = (uint32_t*)calloc(1 << r, sizeof(*histogram));

	#ifdef PRINT_TIMINGS
	std::cout << "Sorting " << (double)(size) / (1024.0 * 1024.0 * 1024.0) << " GB of data" << std::endl;
	#endif

	// populate input
	for (int i = 0; i < count; i++) h_a[i] = rng.Get();

	float cpu_ms = 0.0f;
	{
		int64_t start = GetTimestamp();
		LSDRadixSort(h_a, h_b, count, histogram, r);
		int64_t end = GetTimestamp();
		cpu_ms = GetElapsedMS(start, end);
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "CPU LSD Radix Sort (" << r << " bits keys): " << cpu_ms << " ms" << std::endl;
	#endif

	float gpu_ms = 0.0f;
	{
		hipEvent_t start = MyCudaEventCreate();
		hipEvent_t end = MyCudaEventCreate();
		CUDA_CALL(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
		CUDA_CALL(hipEventRecord(start));
		{
			// 1) Count
			// TODO: initialize smem to zero. 
			// Here smem is used as an histogram. The number of cells in the histogram
			// If histogram is smaller than the block, then some threads won't do any zero initialization
			// If histogram is as big as the block, then each thread will initialize one histogram cell
			// If histogram is bigger than the block, then each thread will initialize one or more histogram cells
			// TODO: build histogram using atomic operations on smem

			// 2) Exclusive Prefix Sum
			// TODO: Compute prefix sum on each block local histogram. This computes the block local offsets
			// TODO: Compute prefix sum on the global histogram, stored in column major order. This computes the global offsets
			// For starters, we write to global memory each block local histogram
			// Then we copy the histogram to another chunk of global memory
			// We perform a block local prefix sum on each histogram in the first chunk of global memory
			// We transpose the second chunk of global memory
			// We perform a prefix sum on the entire second chunk of global memory
			// We transpose again the second chunk of global memory
			// The two prefix sums, can be done on different streams, to help with parallelism

			// 3) Sort
			// TODO: Sort the elements by block, using smem and LSDBinaryRadixSort
			// TODO: Build a destination table using local and global offsets (random access, use SMEM for there tables)
			// Here we require a four times the SMEM. Elements + local offsets + global offsets + destination table
			// TODO: Reorder elements using destination table

			size_t smem = (1 << r) * sizeof(uint32_t);
			BuildHistogramKernel << <grid, block, smem >> > (d_a, count, d_histogram, r, 0);
		}
		CUDA_CALL(hipEventRecord(end));
		CUDA_CALL(hipMemcpy(h_c, d_b, size, hipMemcpyDeviceToHost));
		gpu_ms = MyCudaEventElapsedTime(start, end);
		CUDA_CALL(hipEventDestroy(end));
		CUDA_CALL(hipEventDestroy(start));
	}

	// print timings
	#ifdef PRINT_TIMINGS
	std::cout << "GPU LSD Radix Sort (" << r << " bits keys): " << gpu_ms << " ms - Speedup: x" << cpu_ms / gpu_ms << std::endl;
	#endif

	// TODO: check arrays
	//CheckArrays(h_b, h_c, count);

	// deallocate
	free(histogram);
	CUDA_CALL(hipFree(d_b));
	CUDA_CALL(hipFree(d_a));
	CUDA_CALL(hipHostFree(h_c));
	CUDA_CALL(hipHostFree(h_b));
	CUDA_CALL(hipHostFree(h_a));
	#endif
}

void BenchmarkGPUPrefixSum()
{
	int count[] =
	{
		1024 * 1024 * 64,
		1024 * 1024 * 128,
		1024 * 1024 * 256,
		1024 * 1024 * 512,
		1024 * 1024 * 1024,
	};

	int threads_per_block[] =
	{
		64,
		128,
		256,
		512,
		1024,
	};

	for (int i = 0; i < MYARRAYCOUNT(threads_per_block); i++)
	{
		for (int j = 0; j < MYARRAYCOUNT(count); j++)
		{
			TestGPUPrefixSum(count[i], threads_per_block[j], 0, 10);
		}
	}
}

#define PREFIX_SUM_TEST_ELEMS_COUNT (1024 * 1024)
#define PREFIX_SUM_TEST_ELEMS_THREADS_PER_BLOCK (128)
#define PREFIX_SUM_TEST_ELEMS_MIN 0
#define PREFIX_SUM_TEST_ELEMS_MAX 10

#define BUILD_HISTOGRAM_TEST_ELEMS_COUNT (1024 * 1024)
#define BUILD_HISTOGRAM_TEST_BLOCK_DIM (256)
#define BUILD_HISTOGRAM_TEST_MIN 0
#define BUILD_HISTOGRAM_TEST_MAX 10
#define BUILD_HISTOGRAM_TEST_BIT_GROUP 0
#define BUILD_HISTOGRAM_TEST_R 4

int main()
{
	CheckForHostLeaks();

	#ifdef BENCHMARK_BUILD_HISTOGRAMS
	BenchmarkBuildHistogram();
	#else
	TestSequentialLSDRadixSort();
	TestBlockPrefixSumKernel();
	TestGPUPrefixSum(PREFIX_SUM_TEST_ELEMS_COUNT, PREFIX_SUM_TEST_ELEMS_THREADS_PER_BLOCK, PREFIX_SUM_TEST_ELEMS_MIN, PREFIX_SUM_TEST_ELEMS_MAX);
	TestLSDBinaryRadixSort();
	TestTranspose();
	TestBuildHistogram(BUILD_HISTOGRAM_TEST_ELEMS_COUNT, BUILD_HISTOGRAM_TEST_BLOCK_DIM, BUILD_HISTOGRAM_TEST_R, BUILD_HISTOGRAM_TEST_BIT_GROUP, BUILD_HISTOGRAM_TEST_MIN, BUILD_HISTOGRAM_TEST_MAX);
	TestLSDRadixSort();
	#endif

	return 0;
}
